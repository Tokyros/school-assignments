#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cudaFunctions.h"

__global__ void generateHistograms(int *data, int *hist, int maxIndexToProcess) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	// Handle task overflow
	if (i < maxIndexToProcess) {
		// Increment the current value in the thread's histogram
		hist[data[i] + i * INPUT_MAX_VALUE]++;
	}
}

__global__ void combineHistograms(int *from, int *to, int dataToProceess) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	// Handle task overflow
	if (i < INPUT_MAX_VALUE) {
		// Collect the histograms of each thread into the final histogram
		for (int j = 0; j < dataToProceess; j++) {
			to[i] += from[i + j * INPUT_MAX_VALUE];
		}
	}
}

int computeOnGPU(int *data, int *totalCount, int elementsToProcess, int numberMultThreads){
	
	int CUDAHist[INPUT_MAX_VALUE * elementsToProcess] = {0};

	// Transfare data to GPU
	int *device_data;
	size_t sizeNumberElements = elementsToProcess * sizeof(float);
	hipMalloc((void**) &device_data, sizeNumberElements);
	hipMemcpy(device_data, data, sizeNumberElements, hipMemcpyHostToDevice);
	
	int *device_hist_result;
	size_t sizeNumberRange = INPUT_MAX_VALUE * sizeof(float);
	hipMalloc((void**) &device_hist_result, sizeNumberRange);
	hipMemcpy(device_hist_result, totalCount, sizeNumberRange, hipMemcpyHostToDevice);
	
	int *device_hist_per_thread;
	size_t sizeNumberMultThreads = INPUT_MAX_VALUE * elementsToProcess * sizeof(float);
	hipMalloc((void**) &device_hist_per_thread, sizeNumberMultThreads);
	hipMemcpy(device_hist_per_thread, CUDAHist, sizeNumberMultThreads, hipMemcpyHostToDevice);

	// Generate the histograms per thread
	int threadsPerBlock = 32;
	int blocksPerGrid = (elementsToProcess + threadsPerBlock - 1) / threadsPerBlock;
	generateHistograms<<<blocksPerGrid, threadsPerBlock>>>(device_data, device_hist_per_thread, elementsToProcess);

	// Combine the histograms from each thread into the final result
	threadsPerBlock = 32;
	blocksPerGrid = (INPUT_MAX_VALUE + threadsPerBlock - 1) / threadsPerBlock;
	combineHistograms<<<blocksPerGrid, threadsPerBlock>>>(device_hist_per_thread, device_hist_result, elementsToProcess);

	// Copt data back to host
	hipMemcpy(data, device_data, sizeNumberElements, hipMemcpyDeviceToHost);
	hipMemcpy(totalCount, device_hist_result, sizeNumberRange, hipMemcpyDeviceToHost);
	hipMemcpy(CUDAHist, device_hist_per_thread, sizeNumberMultThreads,hipMemcpyDeviceToHost);

	// Free memory
	hipFree(device_data);
	hipFree(device_hist_result);
	hipFree(device_hist_per_thread);

	return 0;
}