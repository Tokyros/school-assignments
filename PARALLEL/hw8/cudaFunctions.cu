#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__global__ void countDigitViaCuda(int *data, int *countFromCUDA, int numPartElements) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numPartElements) {
		countFromCUDA[data[i] + i * INPUT_MAX_VALUE]++;
	}
}

__global__ void uniteSameDigitsViaCuda(int *countFromCUDA, int *totalCount, int numPartElements) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < INPUT_MAX_VALUE) {
		for (int j = 0; j < PART; j++) {
			totalCount[i] += countFromCUDA[i + j * INPUT_MAX_VALUE];
		}
	}
}

int computeOnGPU(int *data, int *totalCount, int *countFromCUDA, int numPartElements, int rangeNumbers, int numberMultThreads){
	
	hipError_t err = hipSuccess; //Error code to check return values for CUDA calls

	//Define all size_t  
	size_t sizeNumberElements = numPartElements * sizeof(float);
	size_t sizeNumberRange = rangeNumbers * sizeof(float);
	size_t sizeNumberMultThreads = numberMultThreads * sizeof(float);

	//Allocate memory on GPU to copy the data from the host
	int *d_A;
	err = hipMalloc((void**) &d_A, sizeNumberElements);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Allocate memory on GPU to copy the data from the host
	int *d_B;
	err = hipMalloc((void**) &d_B, sizeNumberRange);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Allocate memory on GPU to copy the data from the host
	int *d_C;
	err = hipMalloc((void**) &d_C, sizeNumberMultThreads);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Copy data from host to the GPU memory
	err = hipMemcpy(d_A, data, sizeNumberElements, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Copy data from host to the GPU memory
	err = hipMemcpy(d_B, totalCount, sizeNumberRange, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Copy data from host to the GPU memory
	err = hipMemcpy(d_C, countFromCUDA, sizeNumberMultThreads, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Launch the Kernel to count Digit From Each Process
	int threadsPerBlock = 32;
	int blocksPerGrid = (numPartElements + threadsPerBlock - 1) / threadsPerBlock;
	countDigitViaCuda<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, numPartElements);
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Launch the Kernel to sum the solution from all the threads
	threadsPerBlock = 32;
	blocksPerGrid = (INPUT_MAX_VALUE + threadsPerBlock - 1) / threadsPerBlock;
	uniteSameDigitsViaCuda<<<blocksPerGrid, threadsPerBlock>>>(d_C, d_B, INPUT_MAX_VALUE);
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Copy the result from GPU to the host memory.
	err = hipMemcpy(data, d_A, sizeNumberElements, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Copy the result from GPU to the host memory.
	err = hipMemcpy(totalCount, d_B, sizeNumberRange, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Copy the result from GPU to the host memory.
	err = hipMemcpy(countFromCUDA, d_C, sizeNumberMultThreads,hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Free allocated memory on GPU
	if (hipFree(d_A) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Free allocated memory on GPU
	if (hipFree(d_B) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Free allocated memory on GPU
	if (hipFree(d_C) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	return 0;
}